#include "hip/hip_runtime.h"
#include "../constants.hpp"
#include "../color.hpp"

#include <iostream>
#include <vector>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>


// Can't do std climits 
#define LIMIT 1000000

__constant__ int particleLocations[NUM_PARTICLES][2];

__global__ void cu_varonoi (int *array_d, float *distance) {
	
	/** 
	*	Very small changes to convert the shared memory version
	*	to this version.
	*/
	
	int element = blockIdx.x * BLOCK_SIZE + threadIdx.x;
		
	if (element < (ARRAY_X * ARRAY_Y)) {
		
		/** Set the distance to the max prior to loop. */
		distance[element] = LIMIT;
		__syncthreads();

		for (int i = 0; i < NUM_PARTICLES; ++i) {
			
			/** 
			*	Calculate the euclidean distance. We avoid using powf
			*	here by explicitly multiplying. CUDA best practices
			*	states that powf is an expensive function due to the
			*	necessary checks during computation.
			*	*/
			
			float temp = sqrtf((float) 
				 ((float) (particleLocations[i][0] - (element % ARRAY_X)) * 
				   (particleLocations[i][0] - (element % ARRAY_X))) +
				 ((float) (particleLocations[i][1] - (element / ARRAY_X)) * 
					(particleLocations[i][1] - (element / ARRAY_X))));
			__syncthreads();
			
			if (temp < distance[element]) {
				distance[element] = temp;
				array_d[element] = i;
			}
			__syncthreads();
		
		}
	}

}

int main(int argc, char ** argv) {
		
	int *array = new int[ARRAY_X * ARRAY_Y];
	int *array_d;
	float *distance;
	
	hipError_t result;
	
	int hostParticles[NUM_PARTICLES][2] = { 
		// Removed for brevity
											};
	
	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord(start, 0);
	
	result = hipMalloc ((void**) &array_d, sizeof(int) * (ARRAY_X * ARRAY_Y));
	result = hipMalloc ((void**) &distance, sizeof(float) * (ARRAY_X * ARRAY_Y));
	result = hipMemcpyToSymbol(HIP_SYMBOL(particleLocations), hostParticles, sizeof(int) * 
	                            (2 * NUM_PARTICLES));
	
	if (result != hipSuccess) {
		std::cerr << "hipMalloc (thread) failed." << std::endl;
		delete array;
		exit(1);
	}
		
	dim3 dimblock (BLOCK_SIZE);
	dim3 dimgrid (ceil ((float) (ARRAY_X * ARRAY_Y)/BLOCK_SIZE));
	
	cu_varonoi <<<dimgrid, dimblock>>> (array_d, distance);
	
	result = hipMemcpy (array, array_d, sizeof(int) * (ARRAY_X * ARRAY_Y), 
	                                                udaMemcpyDeviceToHost);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	if (result != hipSuccess) {
		std::cerr << "hipMemcpy host <- dev (thread) failed." << std::endl;
		delete array;
		exit(1);
	}
	
	float elapsedTime;
	hipEventElapsedTime (&elapsedTime, start, stop);
	
	// std::vector<int> out;
	
	// out.insert(out.begin(), std::begin(array), std::end(array));
	
	// printColor (out);
	
	result = hipFree (array_d);
	result = hipFree (distance);
	delete array;
	std::cout << elapsedTime << "\t" << std::endl;

	
	
	return 0;
}